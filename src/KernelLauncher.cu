#include "hip/hip_runtime.h"
#include <cstdlib>
#include <iostream>
#include "KernelLauncher.h"
#include "datatype.h"
#include "DataHolder.h"
// #include "cuPrintf.cu"

//For 1D kernels
#define BLOCKDIM 1024
// For 2D kernels, use 32x32 thread blocks = BLOCKDIM blocks/thread   
// This is the maximum allowable number on my machine, and         
// suffices for coalesced gmem reads.                             
#define BLOCKDIMX 32
#define BLOCKDIMY 32
 
#define NREPS 10

// More trouble than it's worth.
// #define TIMINGINFO(kernelName, griddim, blockdim, ...) do { \
//   hipEventRecord(start); \
//   kernelName##<<<griddim,blockdim>>>(__VA_ARGS__); \
//   hipEventRecord(stop); \
//   hipEventSynchronize(stop); \
//   float ms = 0; \
//   hipEventElapsedTime(&ms, start, stop); \
//   std::cout << "Average elapsed time in " << #kernelName << ":  " << ms << " ms\n"; \
//   std::cout << "Average effective data rate of " << #kernelName ":  " \
//       << dhin.totalElements*sizeof(float)/ms/1e6 << " GB/s\n"; \
// } while(0) 

template<class T> KernelLauncher<T> KernelLauncher<T>::kl;

template<class T> KernelLauncher<T>::KernelLauncher()
{
  hipEventCreate(&start);
  hipEventCreate(&stop);
}

template<class T> KernelLauncher<T>::~KernelLauncher()
{
  hipEventDestroy(start);
  hipEventDestroy(stop);
}

template<class T> void KernelLauncher<T>::startTiming()
{
  hipEventRecord(start);
}

template<class T> void KernelLauncher<T>::finishTiming(const char* kernelName, DataHolder<T>& dh)
{
  hipEventRecord(stop); 
  hipEventSynchronize(stop); 
  float ms = 0; 
  hipEventElapsedTime(&ms, start, stop); 
  std::cout << "sizeof(datatype):  " << sizeof(T) << "\n"; 
  std::cout << "Average elapsed time in " << kernelName << ":  " << ms << " ms\n"; 
  std::cout << "Average effective data rate of " << kernelName << ":  " 
      << dh.totalElements*sizeof(T)/ms/1e6 << " GB/s\n"; 
}

//Kernels

template<class T> __global__ void 
copyKernel(int nx, int ny, T* in, T *out)
{
  int globalxindx = blockIdx.x*blockDim.x + threadIdx.x;
  int globalyindx = blockIdx.y*blockDim.y + threadIdx.y;
  int i = nx*globalyindx+globalxindx; 
  if (globalxindx < nx && globalyindx < ny)
    out[i] = in[i];
}

template<class T> __global__ void 
saxpyKernel(int nx, int ny, T *x, T *y, T a)
{
  // Same structure as other kernels for consistent speed comparison. 
  int globalxindx = blockIdx.x*blockDim.x + threadIdx.x;
  int globalyindx = blockIdx.y*blockDim.y + threadIdx.y;
  int i = nx*globalyindx+globalxindx;
  if (globalxindx < nx && globalyindx < ny) 
    y[i] = a*x[i] + y[i];
  // printf("Thread number %d. y = %f, x = %f\n", threadIdx.x, y[i], x[i]);
}

template<class T> __global__ void 
transposeNaiveKernel(int nx_out /*=ny_in*/
    , int ny_out /*=nx_in*/
    , T* in
    , T* out)
{
  int globalxindx_out = blockIdx.x*blockDim.x + threadIdx.x;
  int globalyindx_out = blockIdx.y*blockDim.y + threadIdx.y;
  /* printf("blockIdx.x %4d threadIdx.x %4d globalxindx %6d\nblockIdx.y %4d threadIdx.y %4d globalyindx %6d\n"
      , blockIdx.x
      , threadIdx.x
      , globalxindx_out
      , blockIdx.y
      , threadIdx.y
      , globalyindx_out); */
  if (globalxindx_out < nx_out && globalyindx_out < ny_out)
    out[nx_out*globalyindx_out+globalxindx_out] = in[ny_out*globalxindx_out+globalyindx_out];
    // out[ny*globalxindx+globalyindx] = in[nx*globalyindx+globalxindx];
}

// Inspired by http://www.cs.nyu.edu/manycores/cuda_many_cores.pdf
template<class T> __global__ void 
transposeFastKernel(int nx_in
    , int ny_in
    , T* in
    , T* out)
{
  __shared__ T staging_in[BLOCKDIMY][BLOCKDIMX];
  __shared__ T staging_out[BLOCKDIMX][BLOCKDIMY];
  // For matrix | A B | with submatrices A, B, C, D, 
  //            | C D |
  // | A B |   = | A_T C_T |
  // | C D |_T   | B_T D_T |  
  // Reads to in_staging are coalesced.
  // Data is transposed within in_staging then written back
  // out to appropriate 32x32 block of out.
  int globalxoffset_in = blockIdx.x*blockDim.x;
  int globalyoffset_in = blockIdx.y*blockDim.y;
  int globalxindx_in = globalxoffset_in + threadIdx.x;
  int globalyindx_in = globalyoffset_in + threadIdx.y; 
  if (globalxindx_in < nx_in && globalyindx_in < ny_in)
    staging_in[threadIdx.y][threadIdx.x] = in[nx_in*globalyindx_in+globalxindx_in];
  __syncthreads();
  if (globalxindx_in < nx_in && globalyindx_in < ny_in)
  {
    staging_out[threadIdx.x][threadIdx.y] = staging_in[threadIdx.y][threadIdx.x];
  }
  __syncthreads();
  if (globalxindx_in < nx_in && globalyindx_in < ny_in)
    out[ny_in*(globalxoffset_in+threadIdx.y)+globalyoffset_in+threadIdx.x] = 
        staging_out[threadIdx.y][threadIdx.x];
}

template<class T> __global__ void 
transposeFastNoBankConfKernel(int nx_in
    , int ny_in
    , T* in
    , T* out)
{
  __shared__ T staging_in[BLOCKDIMY][BLOCKDIMX+1];
  __shared__ T staging_out[BLOCKDIMX][BLOCKDIMY+1];
  // For matrix | A B | with submatrices A, B, C, D, 
  //            | C D |
  // | A B |   = | A_T C_T |
  // | C D |_T   | B_T D_T |  
  // Reads to in_staging are coalesced.
  // Data is transposed within in_staging then written back
  // out to appropriate 32x32 block of out.
  int globalxoffset_in = blockIdx.x*blockDim.x;
  int globalyoffset_in = blockIdx.y*blockDim.y;
  int globalxindx_in = globalxoffset_in + threadIdx.x;
  int globalyindx_in = globalyoffset_in + threadIdx.y;
  if (globalxindx_in < nx_in && globalyindx_in < ny_in)
    staging_in[threadIdx.y][threadIdx.x] = in[nx_in*globalyindx_in+globalxindx_in];
  __syncthreads();
  if (globalxindx_in < nx_in && globalyindx_in < ny_in)
  {
    staging_out[threadIdx.x][threadIdx.y] = staging_in[threadIdx.y][threadIdx.x];
  }
  __syncthreads();
  if (globalxindx_in < nx_in && globalyindx_in < ny_in)
    out[ny_in*(globalxoffset_in+threadIdx.y)+globalyoffset_in+threadIdx.x] =
        staging_out[threadIdx.y][threadIdx.x];
}

template<class T> __global__ void 
matxmatNaiveKernel(int nx
      , int ny
      , T* a
      , T* b
      , T* out)
{
  int globalxindx = blockIdx.x*blockDim.x + threadIdx.x;
  int globalyindx = blockIdx.y*blockDim.y + threadIdx.y;
  int i = nx*globalyindx+globalxindx;
  T sum = 0;
  // #pragma unroll // Unrolling NOT useful here because nx not necessarily known at compile time
  for (int x=0; x<nx; x++)
    if (globalxindx < nx && globalyindx < ny)
      sum += a[nx*globalyindx+x]*b[ny*x+globalxindx];
  if (globalxindx < nx && globalyindx < ny)
    out[i] = sum;
}

template<class T> __global__ void 
matxmatTilesKernel(int nx
      , int ny
      , T* a
      , T* b
      , T* out)
{
  // Square tiles in smem
  __shared__ T tileA[BLOCKDIMX][BLOCKDIMX+1];
  __shared__ T tileB[BLOCKDIMX][BLOCKDIMX+1];

  T sumOut = 0; // Holds output for this thread
  int globalxindx = blockIdx.x*blockDim.x + threadIdx.x;
  int globalyindx = blockIdx.y*blockDim.y + threadIdx.y;
  int tileATopLeftxindx = 0;
  int tileATopLeftyindx = 0;
  int tileBTopLeftxindx = 0;
  int tileBTopLeftyindx = 0;
  bool inRange = (globalxindx < nx && globalyindx < ny);

  for (int tileindx=0; tileindx<(nx+BLOCKDIMX-1)/BLOCKDIMX; tileindx++)
  {
    tileATopLeftxindx = tileindx*BLOCKDIMX;
    tileATopLeftyindx = blockIdx.y*blockDim.y;
    tileBTopLeftxindx = blockIdx.x*blockDim.x;
    tileBTopLeftyindx = tileATopLeftxindx;

    // Load square tiles into smem
    if(inRange)
    {
      // Loads are coalesced for both tileA and tileB.
      tileA[threadIdx.y][threadIdx.x] = a[ny*(tileATopLeftyindx+threadIdx.y)+tileATopLeftxindx+threadIdx.x]; 
      tileB[threadIdx.y][threadIdx.x] = b[nx*(tileBTopLeftyindx+threadIdx.y)+tileBTopLeftxindx+threadIdx.x];
    }

    __syncthreads();

    #pragma unroll // Unrolling could be useful here because BLOCKDIMX known at compile time
    for (int x=0; x<BLOCKDIMX; x++)
      if(inRange)
	sumOut += tileA[threadIdx.y][x]*tileB[x][threadIdx.x];      

    __syncthreads(); 

   // if (nx*globalyindx+globalxindx == 0/*threadIdx.x == 0 && threadIdx.y == 0*/)
   //   printf("blockIdx.x: %d, blockIdx.y: %d, tileindx: %d, tileA[0][1]: %f, tileB[0][1]: %f, \ntileOut[1][0]: %f, globalxindx: %d, globalyindx: %d, # of tileindxs: %d\n\n"
   //       , blockIdx.x
   //       , blockIdx.y
   //       , tileindx
   //       , tileA[0][1]
   //       , tileB[0][1]
   //       , tileOut[1][0]
   //       , globalxindx
   //       , globalyindx
   //       , (nx+BLOCKDIMX-1)/BLOCKDIMX);

  }
  if(inRange)
    out[nx*globalyindx+globalxindx] = sumOut; 
}

template<class T> __global__ void 
reduceYBy2Kernel(int nx
    , int ny
    , int yStride
    , T* inout)
{
  // Reduce like this
  //  a b  ->  a+c b+d  // a+c was produced by thread 1, b+d by thread 2, etc 
  //  c d      garbage  
  //  e f      e+g f+h
  //  g h      garbage
  // Memory accesses are x-contiguous => coalesced 
  // for good choice of blockDim.x
  int globalxindx = blockIdx.x*blockDim.x + threadIdx.x;
  int globalyindx = 2*yStride*(blockIdx.y*blockDim.y + threadIdx.y);

  T sum = 0;

  if (globalxindx < nx && globalyindx < ny)
  {
    sum += inout[nx*globalyindx+globalxindx];
    if ((globalyindx+yStride) < ny)
      sum += inout[nx*(globalyindx+yStride)+globalxindx];
    inout[nx*globalyindx+globalxindx] = sum;
  }
}

// Wrapper functions exposed by KernelLauncher<T>

template<class T> void KernelLauncher<T>::copy(DataHolder<T>& dhin
    , DataHolder<T>& dhout)
{
  startTiming();
  copyKernel<<<dim3((dhin.nx()+BLOCKDIMX-1)/BLOCKDIMX,(dhin.ny()+BLOCKDIMY-1)/BLOCKDIMY) \
      , dim3(BLOCKDIMX,BLOCKDIMY)>>>(dhin.nx()
      , dhin.ny()
      , dhin.rawPtrGPU
      , dhout.rawPtrGPU);
  finishTiming("copyKernel", dhin);
}

template<class T> void KernelLauncher<T>::saxpy(DataHolder<T>& dhx
    , DataHolder<T>& dhy
    , T a)
{
  startTiming();
  // for (int rep=0; rep<NREPS; rep++)
    saxpyKernel
	<<<dim3((dhx.nx()+BLOCKDIMX-1)/BLOCKDIMX,(dhx.ny()+BLOCKDIMY-1)/BLOCKDIMY), dim3(BLOCKDIMX,BLOCKDIMY)>>>
	(dhx.nx()
	, dhx.ny()
	, dhx.rawPtrGPU
	, dhy.rawPtrGPU
	, a);
  finishTiming("saxpy", dhx);
}

template<class T> void KernelLauncher<T>::transposeNaive(DataHolder<T>& dhin
    , DataHolder<T>& dhout)
{
  startTiming(); 
  // for (int rep=0; rep<NREPS; rep++)
    transposeNaiveKernel
	<<<dim3((dhout.nx()+BLOCKDIMX-1)/BLOCKDIMX,(dhout.ny()+BLOCKDIMY-1)/BLOCKDIMY), dim3(BLOCKDIMX,BLOCKDIMY)>>>
	(dhout.nx()
	, dhout.ny()
	, dhin.rawPtrGPU
	, dhout.rawPtrGPU);
  finishTiming("transposeNaive", dhin);
}

template<class T> void KernelLauncher<T>::transposeFast(DataHolder<T>& dhin
    , DataHolder<T>& dhout)
{
  if (BLOCKDIMX != BLOCKDIMY) 
    printf("Warning:  transposeFast will fail if BLOCKDIMX (%d) != BLOCKDIMY (%d)"
        , BLOCKDIMX
        , BLOCKDIMY);
  startTiming();
  // for (int rep=0; rep<NREPS; rep++)
    transposeFastKernel
	<<<dim3((dhin.nx()+BLOCKDIMX-1)/BLOCKDIMX,(dhin.ny()+BLOCKDIMY-1)/BLOCKDIMY), dim3(BLOCKDIMX,BLOCKDIMY)>>>
	(dhin.nx()
	, dhin.ny()
	, dhin.rawPtrGPU
	, dhout.rawPtrGPU);
  finishTiming("transposeFast", dhin);
}

template<class T> void KernelLauncher<T>::transposeFastNoBankConf(DataHolder<T>& dhin
    , DataHolder<T>& dhout)
{
  if (BLOCKDIMX != BLOCKDIMY)
    printf("Warning:  transposeFastNoBankConf will fail if BLOCKDIMX (%d) != BLOCKDIMY (%d)"
        , BLOCKDIMX
        , BLOCKDIMY);
  startTiming();
  // for (int rep=0; rep<NREPS; rep++)
    transposeFastNoBankConfKernel
	<<<dim3((dhin.nx()+BLOCKDIMX-1)/BLOCKDIMX,(dhin.ny()+BLOCKDIMY-1)/BLOCKDIMY), dim3(BLOCKDIMX,BLOCKDIMY)>>>
	(dhin.nx()
	, dhin.ny()
	, dhin.rawPtrGPU
	, dhout.rawPtrGPU);
  finishTiming("transposeFastNoBankConf", dhin);
}

template<class T> void KernelLauncher<T>::transpose32PerThread(DataHolder<T>& dhin, DataHolder<T>& dhout)
{
  printf("Currently just runs transposeFast.  To be done later...");
  startTiming();
  // for (int rep=0; rep<NREPS; rep++)
    transposeFastKernel
	<<<dim3((dhin.nx()+BLOCKDIMX-1)/BLOCKDIMX,(dhin.ny()+BLOCKDIMY-1)/BLOCKDIMY), dim3(BLOCKDIMX,BLOCKDIMY)>>>
	(dhin.nx()
	, dhin.ny()
	, dhin.rawPtrGPU
	, dhout.rawPtrGPU);
  finishTiming("transpose32PerThread", dhin);
}

template<class T> void KernelLauncher<T>::matxmatNaive(DataHolder<T>& dha
    , DataHolder<T>& dhb
    , DataHolder<T>& dhout)
{
  startTiming();
  // for (int rep=0; rep<NREPS; rep++)
    matxmatNaiveKernel
	<<<dim3((dha.nx()+BLOCKDIMX-1)/BLOCKDIMX,(dha.ny()+BLOCKDIMY-1)/BLOCKDIMY), dim3(BLOCKDIMX,BLOCKDIMY)>>>
	(dha.nx()
	, dha.ny()
	, dha.rawPtrGPU
	, dhb.rawPtrGPU
	, dhout.rawPtrGPU);
  finishTiming("matxmatNaive", dha);
}

template<class T> void KernelLauncher<T>::matxmatTiles(DataHolder<T>& dha
    , DataHolder<T>& dhb
    , DataHolder<T>& dhout)
{
  startTiming();
  // Uses square tiles.  
  // printf("Using %dx%d grid of thread blocks\n",(dhb.nx()+BLOCKDIMX-1)/BLOCKDIMX,(dha.ny()+BLOCKDIMX-1)/BLOCKDIMX);
  // for (int rep=0; rep<NREPS; rep++)
    matxmatTilesKernel
	<<<dim3((dhb.nx()+BLOCKDIMX-1)/BLOCKDIMX,(dha.ny()+BLOCKDIMX-1)/BLOCKDIMX), dim3(BLOCKDIMX,BLOCKDIMX)>>>
	(dha.nx()
	, dha.ny()
	, dha.rawPtrGPU
	, dhb.rawPtrGPU
	, dhout.rawPtrGPU);
  finishTiming("matxmatTiles", dha);
}

template<class T> void KernelLauncher<T>::reduceY(DataHolder<T>& dhin
    , DataHolder<T>& dhout)
{
  copyKernel
      <<<dim3((dhin.nx()+BLOCKDIMX-1)/BLOCKDIMX,(dhin.ny()+BLOCKDIMY-1)/BLOCKDIMY), dim3(BLOCKDIMX,BLOCKDIMY)>>>
      // <<<dim3(1,(dhin.totalElements+BLOCKDIM-1)/BLOCKDIM),dim3(BLOCKDIM,1)>>>
      (dhin.nx()
      , dhin.ny()
      , dhin.rawPtrGPU
      , dhout.rawPtrGPU);
  startTiming();
  // for (int rep=0; rep<NREPS; rep++) 
  {
    int yStride = 1;
    int pass = 0;
    int nyRemaining = dhout.ny();
    while (nyRemaining >= 2)
    {
      // std::cout << "pass:  " << pass << std::endl;
      // std::cout << "yStride:  " << yStride << std::endl;
      // std::cout << "nyRemaining:  " << nyRemaining << std::endl;
      // std::cout << "Launching with: " << (nyRemaining+2*BLOCKDIMY-1)/(2*BLOCKDIMY) << " blocks in Y direction " << std::endl;
      reduceYBy2Kernel
	  <<<dim3((dhout.nx()+BLOCKDIMX-1)/BLOCKDIMX,(nyRemaining+2*BLOCKDIMY-1)/(2*BLOCKDIMY)), dim3(BLOCKDIMX,BLOCKDIMY)>>>
	  (dhout.nx()
	   , dhout.ny()
           , yStride
	   , dhout.rawPtrGPU);
      pass++;
      yStride *= 2; 
      nyRemaining /= 2;
    }
  }
  finishTiming("reduceYBy2Kernel", dhin);
}


// Force instantiation of KernelLauncher<> for datatype selected in datatype.h
template class KernelLauncher<datatype>;

